#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <assert.h>
#include <stdio.h>

using half_t = half;
__device__ inline float to_float(const half_t & u) { return __half2float(u); }
__device__ inline half_t to_half(const float & u) { return __float2half_rn(u); }
typedef half_t * __restrict__ F_;
__global__ void forward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, half_t* y_, float* s_, float* sa_) {
    constexpr int C = _C_;
    int bb = blockIdx.y, hh = blockIdx.x, i = threadIdx.x;

    float state[C] = {0};
    __shared__ float q[C], k[C], w[C], a[C], b[C];

    for (int t = 0; t < T; t++) {
        int ind = bb*T*H*C + t*H*C + hh * C + i;
        __syncthreads();
        q[i] = to_float(q_[ind]);
        w[i] = __expf(-__expf(to_float(w_[ind])));
        k[i] = to_float(k_[ind]);
        a[i] = to_float(a_[ind]);
        b[i] = to_float(b_[ind]);
        __syncthreads();

        float sa = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            sa += a[j] * state[j];
        }
        sa_[ind] = sa;

        float v = to_float(v_[ind]);
        float y = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            float& s = state[j];
            s = s * w[j] + sa * b[j] + k[j] * v;
            y += s * q[j];
        }
        y_[ind] = to_half(y);

        if ((t+1)%_CHUNK_LEN_ == 0) {
            int base = (bb*H+hh)*(T/_CHUNK_LEN_)*C*C + (t/_CHUNK_LEN_)*C*C + i;
#pragma unroll
            for (int j = 0; j < C; j++) {
                s_[base + j*C] = state[j];
            }
        }
    }
}
__global__ void backward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ dy_, float * __restrict__ s_, float * __restrict__ sa_, half_t* dw_, half_t* dq_, half_t* dk_, half_t* dv_, half_t* da_, half_t* db_) {
    constexpr int C = _C_;
    int bb = blockIdx.y, hh = blockIdx.x, i = threadIdx.x;

    float stateT[C] = {0}, dstate[C] = {0}, dstateT[C] = {0};
    __shared__ float w[C], q[C], k[C], v[C], a[C], b[C], dy[C], sa[C], dSb_shared[C];
    float qi, wi, ki, ai, bi, dyi;

    for (int t = T-1; t >= 0; t--) {
        int ind = bb*T*H*C + t*H*C + hh * C + i;
        __syncthreads();
        q[i] = qi = to_float(q_[ind]);
        float wi_fac = -__expf(to_float(w_[ind]));
        w[i] = wi = __expf(wi_fac);
        k[i] = ki = to_float(k_[ind]);
        a[i] = ai = to_float(a_[ind]);
        b[i] = bi = to_float(b_[ind]);
        v[i] = to_float(v_[ind]);
        dy[i] = dyi = to_float(dy_[ind]);
        sa[i] = sa_[ind];
        __syncthreads();

        if ((t+1)%_CHUNK_LEN_ == 0) {
            int base = (bb*H+hh)*(T/_CHUNK_LEN_)*C*C + (t/_CHUNK_LEN_)*C*C + i*C;
#pragma unroll
            for (int j = 0; j < C; j++) {
                stateT[j] = s_[base + j];
            }
        }

        float dq = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            dq += stateT[j]*dy[j];
        }
        dq_[ind] = to_half(dq);

        float iwi = 1.0f/wi;
#pragma unroll        
        for (int j = 0; j < C; j++) {
            stateT[j] = (stateT[j] - ki*v[j] - bi*sa[j]) * iwi;
            dstate[j] += dyi * q[j];
            dstateT[j] += qi * dy[j];
        }

        float dw = 0, dk = 0, dv = 0, db = 0, dSb = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            dw += dstateT[j]*stateT[j];
            dk += dstateT[j]*v[j];
            dv += dstate[j]*k[j];
            dSb += dstate[j]*b[j];
            db += dstateT[j]*sa[j];
        }
        dw_[ind] = to_half(dw * wi * wi_fac);
        dk_[ind] = to_half(dk);
        dv_[ind] = to_half(dv);
        db_[ind] = to_half(db);

        __syncthreads();
        dSb_shared[i] = dSb;
        __syncthreads();

        float da = 0;
#pragma unroll
        for (int j = 0; j < C; j++) {
            da += stateT[j]*dSb_shared[j];
        }
        da_[ind] = to_half(da);

#pragma unroll        
        for (int j = 0; j < C; j++) {
            dstate[j] = dstate[j]*w[j] + dSb * a[j];
            dstateT[j] = dstateT[j]*wi + ai * dSb_shared[j];
        }
    }
}
void cuda_forward(int B, int T, int H, half_t*w, half_t*q, half_t*k, half_t*v, half_t*z, half_t*a, half_t*y, float*s, float*sa) {
    forward_kernel<<<dim3(H,B), dim3(_C_)>>>(T,H,w,q,k,v,z,a,y,s,sa);
}
void cuda_backward(int B, int T, int H, half_t*w, half_t*q, half_t*k, half_t*v, half_t*z, half_t*a, half_t*dy, float*s, float*sa, half_t*dw, half_t*dq, half_t*dk, half_t*dv, half_t*dz, half_t*da) {
    assert(T%_CHUNK_LEN_ == 0);
    backward_kernel<<<dim3(H,B), dim3(_C_)>>>(T,H,w,q,k,v,z,a,dy,s,sa,dw,dq,dk,dv,dz,da);
}